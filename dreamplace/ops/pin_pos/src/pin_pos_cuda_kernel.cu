#include "hip/hip_runtime.h"
#include <cfloat>
#include <stdio.h>
#include "assert.h"
#include "hip/hip_runtime.h"
#include "utility/src/print.h"
#include "utility/src/Msg.h"
#include <hipcub/hipcub.hpp>

DREAMPLACE_BEGIN_NAMESPACE

template <typename T>
__global__ void permuteGrad(
	const T* grad_out_x,
	const T* grad_out_y,
	const int* flat_node2pin_map,
	const int num_pins,
	T* grad_out_x_perm,
	T* grad_out_y_perm
	)
{
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < num_pins; i += blockDim.x * gridDim.x)
    {
        int pin_id = flat_node2pin_map[i];
	grad_out_x_perm[i] = grad_out_x[pin_id];
	grad_out_y_perm[i] = grad_out_y[pin_id];
    }
}

template <typename T>
void sortByKey(
	const long* old_keys, 
	long* keys_sorted, 
	const T* array_unsorted, 
	T* array_sorted, 
	int array_size
	)
{
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, 
	    old_keys, keys_sorted, array_unsorted, array_sorted, array_size, 0, sizeof(int)*8);

    hipMalloc(&d_temp_storage, temp_storage_bytes);

    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
	    old_keys, keys_sorted, array_unsorted, array_sorted, array_size, 0, sizeof(int)*8);

    hipFree(d_temp_storage);
}

struct CustomAdd
{
    template <typename T>
    HIPCUB_RUNTIME_FUNCTION __forceinline__
    T operator()(const T &a, const T &b) const{
	return a + b;
    }    
};

template <typename T>
void reduceByKey(
	const long* keys,
	const T* vals,
	T* sum_reduced,
	int num_val
	)
{
    int *d_unique_out;
    hipMalloc((void**)&d_unique_out, num_val*sizeof(int));
    int *d_num_runs_out;
    hipMalloc((void**)&d_num_runs_out, sizeof(int));
    CustomAdd reduction_op;

    long *keys_sorted;
    hipMalloc((void**)&keys_sorted, num_val*sizeof(long));
    T *vals_sorted;
    hipMalloc((void**)&vals_sorted, num_val*sizeof(T));

    sortByKey(keys, keys_sorted, vals, vals_sorted, num_val);

    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::ReduceByKey(d_temp_storage, temp_storage_bytes, keys_sorted, d_unique_out, 
	    vals_sorted, sum_reduced, d_num_runs_out, reduction_op, num_val);

    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceReduce::ReduceByKey(d_temp_storage, temp_storage_bytes, keys_sorted, d_unique_out,
	    vals_sorted, sum_reduced, d_num_runs_out, reduction_op, num_val);

    hipFree(d_unique_out);
    hipFree(d_num_runs_out);
    hipFree(keys_sorted);
    hipFree(vals_sorted);
    hipFree(d_temp_storage);
}

template <typename T>
void segmentSum(
	const T* d_in,
	T* d_out,
	const int* d_offset,
	const int num_seg
	)
{
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out, 
	    num_seg, d_offset, d_offset + 1);

    hipMalloc(&d_temp_storage, temp_storage_bytes);

    hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out,
	    num_seg, d_offset, d_offset + 1);

    hipFree(d_temp_storage);
}

template <typename T>
int computePinPosCudaLauncher(
	const T* x, const T* y,
	const T* pin_offset_x,
	const T* pin_offset_y,
	const long* pin2node_map,
	const int* flat_node2pin_map,
	const int* flat_node2pin_start_map,
	int num_pins,
	T* pin_x, T* pin_y
    )
{
    hipMemcpy(pin_x, pin_offset_x, num_pins * sizeof(T), hipMemcpyDefault);
    hipMemcpy(pin_y, pin_offset_y, num_pins * sizeof(T), hipMemcpyDefault);
    reduceByKey(pin2node_map, x, pin_x, num_pins);
    reduceByKey(pin2node_map, y, pin_y, num_pins);

    return 0;
}

template <typename T>
int computePinPosGradCudaLauncher(
	const T* grad_out_x, const T* grad_out_y,
	const T* x, const T* y,
	const T* pin_offset_x,
	const T* pin_offset_y,
	const long* pin2node_map,
	const int* flat_node2pin_map,
	const int* flat_node2pin_start_map,
	int num_nodes,
	int num_pins,
	T* grad_x, T* grad_y
    )
{
    int thread_count = 1024;
    int block_count = 32;

    T *grad_out_x_perm, *grad_out_y_perm;
    hipMalloc((void**)&grad_out_x_perm, num_pins*sizeof(T));
    hipMalloc((void**)&grad_out_y_perm, num_pins*sizeof(T));

    permuteGrad<<<block_count, thread_count>>>(grad_out_x, grad_out_y, flat_node2pin_map, num_pins, grad_out_x_perm, grad_out_y_perm);

    segmentSum(grad_out_x_perm, grad_x, flat_node2pin_start_map, num_nodes);
    segmentSum(grad_out_y_perm, grad_y, flat_node2pin_start_map, num_nodes);

    hipFree(grad_out_x_perm);
    hipFree(grad_out_y_perm);
    return 0;	
}


#define REGISTER_KERNEL_LAUNCHER(T) \
    int instantiateComputePinPosCudaLauncher(\
    	    const T* x, const T* y, \
    	    const T* pin_offset_x, \
	        const T* pin_offset_y, \
	        const long* pin2node_map, \
	        const int* flat_node2pin_map, \
	        const int* flat_node2pin_start_map, \
	        int num_pins, \
	        T* pin_x, T* pin_y \
            )\
    {\
        return computePinPosCudaLauncher(\
    	        x, y, \
    	        pin_offset_x, \
	            pin_offset_y, \
	            pin2node_map, \
	            flat_node2pin_map, \
	            flat_node2pin_start_map, \
	            num_pins, \
	            pin_x, pin_y \
                );\
    } \
    \
    int instantiateComputePinPosGradCudaLauncher(\
        	const T* grad_out_x, const T* grad_out_y, \
	        const T* x, const T* y, \
	        const T* pin_offset_x, \
	        const T* pin_offset_y, \
	        const long* pin2node_map, \
	        const int* flat_node2pin_map, \
	        const int* flat_node2pin_start_map, \
	        int num_nodes, \
	        int num_pins, \
	        T* grad_x, T* grad_y \
            )\
    {\
        return computePinPosGradCudaLauncher(\
        	    grad_out_x, grad_out_y, \
	            x, y, \
	            pin_offset_x, \
	            pin_offset_y, \
	            pin2node_map, \
	            flat_node2pin_map, \
	            flat_node2pin_start_map, \
	            num_nodes, \
	            num_pins, \
	            grad_x, grad_y \
                );\
    }
REGISTER_KERNEL_LAUNCHER(float);
REGISTER_KERNEL_LAUNCHER(double);

DREAMPLACE_END_NAMESPACE