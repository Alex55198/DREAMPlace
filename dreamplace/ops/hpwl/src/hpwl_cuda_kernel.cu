#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <float.h>
#include "hip/hip_runtime.h"
#include "utility/src/Msg.h"

DREAMPLACE_BEGIN_NAMESPACE

template <typename T>
void printArray(const T* x, const int n, const char* str)
{
    printf("%s[%d] = ", str, n); 
    T* host_x = (T*)malloc(n*sizeof(T));
    if (host_x == NULL)
    {
        printf("failed to allocate memory on CPU\n");
        return;
    }
    hipMemcpy(host_x, x, n*sizeof(T), hipMemcpyDeviceToHost);
    for (int i = 0; i < n; ++i)
    {
        printf("%g ", double(host_x[i]));
    }
    printf("\n");

    free(host_x);
}

template <typename T>
void printScalar(const T& x, const char* str)
{
    printf("%s = ", str); 
    T* host_x = (T*)malloc(sizeof(T));
    if (host_x == NULL)
    {
        printf("failed to allocate memory on CPU\n");
        return;
    }
    hipMemcpy(host_x, &x, sizeof(T), hipMemcpyDeviceToHost);
    printf("%g\n", double(*host_x));

    free(host_x);
}

template <typename T>
__global__ void fillArray(T* x, const int n, const T v)
{
    //for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) 
    int i  = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        x[i] = v; 
    }
}

template <typename T>
__global__ void computeHPWL(
        const T* x, const T* y, 
        const int* flat_netpin, 
        const int* netpin_start, 
        const unsigned char* net_mask, 
        int num_nets,
        T* partial_hpwl 
        )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_nets)
    {
        T max_x = -FLT_MAX;
        T min_x = FLT_MAX;
        T max_y = -FLT_MAX;
        T min_y = FLT_MAX;

        if (net_mask[i])
        {
            for (int j = netpin_start[i]; j < netpin_start[i+1]; ++j)
            {
                int k = flat_netpin[j];
                T xx = x[k]; 
                T yy = y[k]; 
                min_x = min(min_x, xx);
                max_x = max(max_x, xx);
                min_y = min(min_y, yy);
                max_y = max(max_y, yy);
            }
            partial_hpwl[i] = max_x-min_x; 
            partial_hpwl[i+num_nets] = max_y-min_y; 
        }
        else 
        {
            partial_hpwl[i] = 0; 
            partial_hpwl[i+num_nets] = 0; 
        }
    }
}

template <typename T>
int computeHPWLCudaLauncher(
        const T* x, const T* y, 
        const int* flat_netpin, 
        const int* netpin_start, 
        const unsigned char* net_mask, 
        int num_nets,
        T* partial_hpwl
        )
{
    const int thread_count = 512; 

    computeHPWL<<<(num_nets+thread_count-1) / thread_count, thread_count>>>(
            x, y,
            flat_netpin, 
            netpin_start, 
            net_mask, 
            num_nets,
            partial_hpwl
            );

    //printArray(partial_hpwl, num_nets, "partial_hpwl");

    // I move out the summation to use ATen 
    // significant speedup is observed 
    //sumArray<<<1, 1>>>(partial_hpwl, num_nets, hpwl);

    return 0; 
}

// manually instantiate the template function 
#define REGISTER_KERNEL_LAUNCHER(type) \
    int instantiateComputeHPWLLauncher(\
        const type* x, const type* y, \
        const int* flat_netpin, \
        const int* netpin_start, \
        const unsigned char* net_mask, \
        int num_nets, \
        type* partial_hpwl \
        ) \
    { \
        return computeHPWLCudaLauncher(x, y, \
                flat_netpin, \
                netpin_start, \
                net_mask, \
                num_nets, \
                partial_hpwl \
                ); \
    }

REGISTER_KERNEL_LAUNCHER(float);
REGISTER_KERNEL_LAUNCHER(double);

DREAMPLACE_END_NAMESPACE
